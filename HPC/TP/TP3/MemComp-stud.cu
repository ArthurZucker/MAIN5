/**************************************************************
This code compares standard CPU allocation with the locked one.
It also checks the effeciency of mapping the CPU memory 

This code is a part of a course on cuda taught by the author: 
Lokman A. Abbas-Turki

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

// This kernel is needed to compare the mapped memory to other memories
__global__ void test_kernel(int *Tab, int size, int i){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x<size){
		Tab[x] = i;
	}
}

float malloc_trans(int size, int NbT, bool flag) {

	int *a, *aGPU;
	float TimeVar;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));

	a = (int*)malloc(size*sizeof(int));
	testCUDA(hipMalloc(&aGPU,size*sizeof(int)));

	testCUDA(hipEventRecord(start,0));

	for (int i=0; i<NbT; i++) {
		if (flag){
			testCUDA(hipMemcpy(aGPU, a, size*sizeof(int),	hipMemcpyHostToDevice)); 
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
		}else{
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
			testCUDA(hipMemcpy(a, aGPU, size*sizeof(int),	hipMemcpyDeviceToHost));
		}
	}

	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipFree(aGPU));
	free(a);	
	return TimeVar;
}


float malloc_host(int size, int NbT, bool flag) {

	int *a, *aGPU;
	float TimeVar;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
	testCUDA(hipHostAlloc(&a,size*sizeof(int),hipHostMallocDefault));
	testCUDA(hipMalloc(&aGPU,size*sizeof(int)));
	testCUDA(hipEventRecord(start,0));

	for (int i=0; i<NbT; i++) {
		if (flag){
			testCUDA(hipMemcpy(aGPU, a, size*sizeof(int),	hipMemcpyHostToDevice)); 
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
		}else{
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
			testCUDA(hipMemcpy(a, aGPU, size*sizeof(int),	hipMemcpyDeviceToHost));
		}
	}

	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipHostFree(a));
	testCUDA(hipFree(aGPU));
	return TimeVar;
}

float malloc_map(int size, int NbT, bool flag) {

	int *a, *aGPU;
	float TimeVar;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
	testCUDA(hipHostAlloc(&a,size*sizeof(int),hipHostMallocMapped));
	testCUDA(hipHostGetDevicePointer((void **)&aGPU, (void *) a, 0));
	testCUDA(hipEventRecord(start,0));
	for (int i=0; i<NbT; i++) {
		if (flag){
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
		}else{
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
			
		}
	}
	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipHostFree(a));
	return TimeVar;
}

int main (void){

	int size = 1024*1024;
	int NbT = 1000;
	float TimeVar;

	testCUDA(hipSetDeviceFlags(hipDeviceMapHost));

	TimeVar = malloc_trans(size, NbT, true);
	printf("Processing time when using malloc CPU2GPU\t: %f s\n", 0.001f*TimeVar);
	TimeVar = malloc_trans(size, NbT, false);
	printf("Processing time when using malloc GPU2CPU\t: %f s\n", 0.001f*TimeVar);
	TimeVar = malloc_host(size, NbT, true);
	printf("Processing time when using mallocHOST CPU2GPU\t: %f s\n", 0.001f*TimeVar);
	TimeVar = malloc_host(size, NbT, false);
	printf("Processing time when using mallocHOST GPU2CPU\t: %f s\n", 0.001f*TimeVar);

	TimeVar = malloc_map(size, NbT, true);
	printf("Processing time when using mallocMAP CPU2GPU\t: %f s\n", 0.001f*TimeVar);
	TimeVar = malloc_map(size, NbT, false);
	printf("Processing time when using mallocMAP GPU2CPU\t: %f s\n", 0.001f*TimeVar);
    // TODO: do the same with locked and mapped memories

	return 0;
}
