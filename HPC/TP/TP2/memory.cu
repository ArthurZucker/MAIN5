
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define PINMEMORY  0


// vecteur en mémoire globale
__device__   float b[7]         = {0.2316419f,0.3193815f,-0.3565638f,1.781478f,-1.821256f,1.330274f,0.3989423f};
__constant__ float constant_b[7]= {0.2316419f,0.3193815f,-0.3565638f,1.781478f,-1.821256f,1.330274f,0.3989423f};
__shared__ float shared_b[7];
__device__ float NP_r(float x);
__device__ float NP_g(float x);
__device__ float NP_s(float x);
__device__ float NP_c(float x);

__global__ void NP_register     (float *x, float *y, int N);
__global__ void NP_global       (float *x, float *y, int N);
__global__ void NP_constant     (float *x, float *y, int N);
__global__ void NP_global2      (float *x, float *y, int N);
__global__ void NP_shared       (float *x, float *y, int N);
// __global__ void NP_shared2      (float *x, float *y, int N);       

int main() {
  srand((unsigned int)time(NULL));
  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
    }
  
  hipSetDevice(0);


  FILE *fptr=NULL;

  for (int k=2;k<1679000;k+=1000){
    int N = k;
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    size_t size = N * sizeof(float);

    // Allocate input vectors x and y in host memory
    float* x    = (float*)malloc(size);
    float* yS   = (float*)malloc(size);
    float* yS2  = (float*)malloc(size);
    float* yR   = (float*)malloc(size);
    float* yG   = (float*)malloc(size);
    float* yG2  = (float*)malloc(size);
    float* yC   = (float*)malloc(size);
    // Initialize input vectors
    for (int i =0; i<N;i++){
        x[i] =  (float)rand();
    }

    // Allocate vectors in device memory
    float* d_xg;
    float* d_yg;
    float* d_xr;
    float* d_yr;
    float* d_xs;
    float* d_ys;
    float* d_xc;
    float* d_yc;
    float* d_xs2;
    float* d_ys2;
    float* d_xg2;
    float* d_yg2;

    // CUDA malloc, page lock memory + pin memory 
    // global
    hipMalloc(&d_xg, size);
    hipMalloc(&d_yg, size);
    // global2
    hipMalloc(&d_xg2, size);
    hipMalloc(&d_yg2, size);
    // register
    hipMalloc(&d_xr, size);
    hipMalloc(&d_yr, size);
    // shared
    hipMalloc(&d_xs, size);
    hipMalloc(&d_ys, size);
    // shared2
    hipMalloc(&d_xs2, size);
    hipMalloc(&d_ys2, size);
    // constant
    hipMalloc(&d_xc, size);
    hipMalloc(&d_yc, size);
    
    #if PINMEMORY == 1
    cudaHostRegister(&d_xg2, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_xg, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_yg, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_yg2, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_xs, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_yr, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_xr, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_ys, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_xs2, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_ys2, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_yc, size,cudaHostRegisterDefault);
    cudaHostRegister(&d_xc, size,cudaHostRegisterDefault);
    #endif
    // Time measurment
    hipEvent_t startR, stopR,startcp,stopcp,startG,stopG,startS,stopS,startC,stopC,startG2,stopG2,startS2,stopS2;
    hipEventCreate(&startR);
    hipEventCreate(&stopR);
    hipEventCreate(&startcp);
    hipEventCreate(&stopcp);
    hipEventCreate(&startG);
    hipEventCreate(&stopG);
    hipEventCreate(&startS);
    hipEventCreate(&stopS);
    hipEventCreate(&startC);
    hipEventCreate(&stopC);
    hipEventCreate(&startG2);
    hipEventCreate(&stopG2);
    hipEventCreate(&startS2);
    hipEventCreate(&stopS2);
    hipEventRecord(startcp);
    // measure global time taken 

    // Copy vectors from host memory to device memory
    hipMemcpy(d_xg, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_xr, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_xs, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_xc, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_xg2, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_xs2, x, size, hipMemcpyHostToDevice);
    //-----------************ Kernel usage ************-----------//

    // Using registers
    hipEventRecord(startR);
    NP_register<<<blocksPerGrid, threadsPerBlock>>>(d_xr, d_yr, N);
    hipEventRecord(stopR);
    // End registers
    hipMemcpy(yR, d_yr, size, hipMemcpyDeviceToHost);              // first result, useless?

    
    // Using global
    hipEventRecord(startG);
    NP_global<<<blocksPerGrid, threadsPerBlock>>>(d_xg, d_yg, N);
    hipEventRecord(stopG);
    // End global
    hipMemcpy(yG, d_yg, size, hipMemcpyDeviceToHost);

    // Using global2
    hipEventRecord(startG2);
    NP_global2<<<blocksPerGrid, threadsPerBlock>>>(d_xg2, d_yg2, N);
    hipEventRecord(stopG2);
    // End global2
    hipMemcpy(yG2, d_yg2, size, hipMemcpyDeviceToHost);

    // Using shared
    hipEventRecord(startS);
    NP_shared<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_ys, N);
    hipEventRecord(stopS);
    // End shared
    hipMemcpy(yS, d_ys, size, hipMemcpyDeviceToHost);

//     // Using shared2
//     cudaEventRecord(startS2);
//     NP_shared2<<<blocksPerGrid, threadsPerBlock>>>(d_xs2, d_ys2, N);
//     cudaEventRecord(stopS2);
//     // End shared2
//     cudaMemcpy(yS2, d_ys2, size, cudaMemcpyDeviceToHost); 

    // Using constant
    hipEventRecord(startC);
    NP_constant<<<blocksPerGrid, threadsPerBlock>>>(d_xc, d_yc, N);
    hipEventRecord(stopC);
    // End constant
    hipMemcpy(yC, d_yc, size, hipMemcpyDeviceToHost);

    hipEventRecord(stopcp);
    // End global time 


    hipEventSynchronize(stopR);
    hipEventSynchronize(stopcp);
    hipEventSynchronize(stopG);
    hipEventSynchronize(stopS);
    hipEventSynchronize(stopC);
    //-----------************ End Kernel usage **********----------//

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startR, stopR);

    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, startcp, stopcp);

    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, startG, stopG);

    float milliseconds4 = 0;
    hipEventElapsedTime(&milliseconds4, startS, stopS);

    float milliseconds5 = 0;
    hipEventElapsedTime(&milliseconds5, startC, stopC);

    float milliseconds6 = 0;
    hipEventElapsedTime(&milliseconds6, startG2, stopG2);

//     float milliseconds6 = 0;
//     cudaEventElapsedTime(&milliseconds6, startS2, stopS2);

    // File writing
    fptr = fopen("results.txt","a");
    if(fptr == NULL){
        printf("Error in file opening!");   
        exit(1);             
    }
    fprintf(fptr,"%f,%f,%f,%f,%f\n",milliseconds,milliseconds3,milliseconds4,milliseconds5,milliseconds6);
    fclose(fptr);
    // End file writing
    printf("Iteration : %d\n",k-2);
    printf("Elapsed time : %f ms\n",milliseconds2);
    printf("Elapsed time : %.2f s\n" ,milliseconds2/1000);

//     printf("Checking answers %%\n");
//     for(int i=0;i<N;i++){
//             if(yS[i]!=yG[i] || yC[i]!=yR[i] || yC[i]!=yS[i] || yS[i]!=yR[i] || yG[i]!=yR[i]) {
//                 printf("Some answeres were wrong");
//                 exit(0);
//             }
//     }
//     printf("Done. Answers are correct %%\n");
    // Free device memory
    hipFree(d_xg);
    hipFree(d_yg);
    hipFree(d_xr);
    hipFree(d_yr);
  }
}

__device__ float NP_r(float x ){
        float p = 0.2316419f; 
        float b1 = 0.3193815f; 
        float b2 = -0.3565638f; 
        float b3 = 1.781478f; 
        float b4 = -1.821256f; 
        float b5 = 1.330274f; 
        float one_over_twopi = 0.3989423f; 
        float t; 
        if(x>=0.0f){
                t = 1.0f / ( 1.0f + p * x);
                return (1.0f - one_over_twopi * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*b5+b4)+b3)+b2)+b1) );
        }
        else {
            t = 1.0f /( 1.0f -p *x);
            return (one_over_twopi * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*b5+b4)+b3)+b2)+b1));
        }
}
      
      
__device__ float NP_g(float x){
        float t; 
        if(x>=0.0f){
                t = 1.0f / ( 1.0f + b[0] * x);
                return (1.0f - b[6] * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*b[5]+b[4])+b[3])+b[2])+b[1]) );
        }
        else {
            t = 1.0f /( 1.0f -b[0] *x);
            return (b[6] * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*b[5]+b[4])+b[3])+b[2])+b[1]));
        }
}
      
__device__ float NP_s(float x){
              float t; 
              if(x>=0.0f){
                      t = 1.0f / ( 1.0f + shared_b[0] * x);
                      return (1.0f - shared_b[6] * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*shared_b[5]+shared_b[4])+shared_b[3])+shared_b[2])+shared_b[1]) );
              }
              else {
                  t = 1.0f /( 1.0f -shared_b[0] *x);
                  return (shared_b[6] * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*shared_b[5]+shared_b[4])+shared_b[3])+shared_b[2])+shared_b[1]));
              }
}
  
__device__ float NP_c(float x){
        float t; 
        if(x>=0.0f){
                t = 1.0f / ( 1.0f + constant_b[0] * x);
                return (1.0f - constant_b[6] * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*constant_b[5]+constant_b[4])+constant_b[3])+constant_b[2])+constant_b[1]) );
        }
        else {
            t = 1.0f /( 1.0f -constant_b[0] *x);
            return (constant_b[6] * expf(-x * x / 2.0f) * t * (t*(t*(t*(t*constant_b[5]+constant_b[4])+constant_b[3])+constant_b[2])+constant_b[1]) );
        }
}


__global__ void NP_register(float *x, float *y, int N){
          int i = blockDim.x * blockIdx.x + threadIdx.x;
          if (i < N)
            y[i] = NP_r(x[i]);
}
      
__global__ void NP_global(float *x,float *y,int N){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) y[i] = NP_g(x[i]);
}
      
      
__global__ void NP_constant(float *x,float *y,int N){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) y[i] = NP_c(x[i]);         
}
      
__global__ void NP_shared(float *x,float *y,int N){
              int i = blockDim.x * blockIdx.x + threadIdx.x;
              if (i < 7) shared_b[i] = b[i];
              __syncthreads(); //wait for the 7 threads to have loaded in the shared memory 
              if (i < N) y[i] = NP_s(x[i]);
}
/*__global__ void NP_shared2(float *x,float *y,int N){
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
        pipeline pipe;
        if (i < 7){
                memcpy_async(shared_b[i], b[i], pipe ); // Async-Copy Dispatch
                pipe.commit_and_wait(); //wait for the 7 threads to have loaded in the shared memory
        } 
        block.sync();
        if (i < N) y[i] = NP_s(x[i]);
        block.sync();
}*/

__global__ void NP_global2(float *x,float *y,int N){
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N) y[i] = NP_g(__ldg(&x[i]));         
      }