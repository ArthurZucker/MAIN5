#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <time.h>


__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}


int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
  
  hipSetDevice(0);

  int N = 300000;
  size_t size = N * sizeof(float);
  // Allocate input vectors h_A and h_B in host memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);
  // Initialize input vectors
  for (int i =0; i<N;i++){
      h_A[i] =  i;
      h_B[i] = -i;
  }

  // Allocate vectors in device memory
  float* d_A;
  hipMalloc(&d_A, size);
  float* d_B;
  hipMalloc(&d_B, size);
  float* d_C;
  hipMalloc(&d_C, size);



  hipEvent_t start, stop,startcp,stopcp;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&startcp);
  hipEventCreate(&stopcp);



  clock_t begin_cp = clock();
  hipEventRecord(startcp);
  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  clock_t begin = clock();
  hipEventRecord(start);

  // Kernel
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  hipEventRecord(stop);
  clock_t end = clock();

  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  
  hipEventRecord(stopcp);
  clock_t end_cp = clock();
  double time_spent2 = (double)(end_cp - begin_cp) / CLOCKS_PER_SEC;

  hipEventSynchronize(stop);
  hipEventSynchronize(stopcp);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float milliseconds2 = 0;
  hipEventElapsedTime(&milliseconds2, startcp, stopcp);

  printf("elapsed time no\t cp GPU: %f\n",milliseconds);
  printf("elapsed time no\t cp CPU: %f\n",time_spent);

  printf("elapsed time with cp GPU: %f\n",milliseconds2);
  printf("elapsed time with cp CPU: %f",time_spent2);
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}